/** \file
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <time.h>
#include <sys/time.h>
#include <stdint.h>

int nextPower(int);
void die(const char *);
void warn(const char *);
void read_from_file(int *, char *, int);
void write_to_file(int *, char *, int);

/**
 * play - Plays the game for one step.
 * First, counts the neighbors, taking into account boundary conditions
 * Then, acts on the rules.
 * Updates need to happen all together, so a temporary new array is allocated
 */
__global__ void play(int *X, int *d_new, int N){
    int i = (blockIdx.x*blockDim.x)+threadIdx.x;
    int j = (blockIdx.y*blockDim.y)+threadIdx.y;
    int up, down, left, right;

        if( i<N && j<N){

            int sum = 0;
            // Code below is faster but hard to read
            up = ((i-1)+N)%N;
            down = (i+1)%N;
            left = ((j-1)+N)%N;
            right = (j+1)%N;
            sum =
                X[N*up+left]+   //i-1, j-1
                X[N*up+j]+    //i-1, j
                X[N*up+right]+ //i-1, j+1

                X[N*i+left]+      //i, j-1
                X[N*i+right]+    //i, j+1

                X[N*down+left]+  //i+1, j-1
                X[N*down+j]+   //i+1, j
                X[N*down+right];//i+1, j+1


            //act based on rules
            if(X[i*N+j] == 0  && sum == 3 ){
                d_new[i*N+j]=1; //born
            }else if ( X[i*N+j] == 1  && (sum < 2 || sum>3 ) ){
                d_new[i*N+j]=0; //dies - loneliness or overpopulation
            }else{
                d_new[i*N+j] = X[i*N+j]; //nothing changes
            }
        }
    return;
}
/**
 * main - plays the game of life for t steps according to the rules:
 * - A dead(0) cell with exactly 3 living neighbors becomes alive (birth)
 * - A dead(0) cell with any other number of neighbors stays dead (barren)
 * - A live(1) cell with 0 or 1 living neighbors dies (loneliness)
 * - A live(1) cell with 4 or more living neighbors dies (overpopulation)
 * - A live(1) cell with 2 or 3 living neighbors stays alive (survival)
 */
int main(int argc, char **argv){

    //sanity check for input
    if(argc !=5){
        printf("Usage: %s filename size t threads, where:\n", argv[0]);
        printf("\tfilename is the input file \n");
        printf("\tsize is the grid side and \n");
        printf("\tt generations to play\n");
        printf("\t threadsXthreads per block\n");
        die("Wrong arguments");
    }

    //declarations
    char *filename = argv[1];
    int N = atoi(argv[2]);
    int t = atoi(argv[3]);
    int thrds = atoi(argv[4]);
    int gen = 0;
    int *table = (int *)malloc(N*N*sizeof(int));
    if (!table)
        die("Couldn't allocate memory to table");

    //read input
    read_from_file(table, filename, N);

    //get the smallest power of 2 larger than N
    int Npow2 = nextPower(N);

    //CUDA - timing
    float gputime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //CUDA - split board into squares 
    dim3 threadsPerBlock(thrds, thrds);
    dim3 numBlocks(Npow2/threadsPerBlock.x, Npow2/threadsPerBlock.y);

    //CUDA - copy input to device
    int *d_table;
    hipMalloc(&d_table, N*N*sizeof(int));
    int *d_new;
    hipMalloc(&d_new, N*N*sizeof(int));
    hipMemcpy(d_table, table, N*N*sizeof(int), hipMemcpyHostToDevice);

    //CUDA - play game for t generations
    hipEventRecord(start, 0);
    for(gen=0; gen<t; gen++){
        //alternate between using d_table and d_new as temp
        if(gen%2==0){
            play<<<numBlocks, threadsPerBlock>>>(d_table /*data*/, d_new /*temp*/, N);
        }else{
            play<<<numBlocks, threadsPerBlock>>>(d_new /*data*/, d_table /*temp*/, N);
        }
        hipDeviceSynchronize(); //don't continue if kernel not done
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gputime, start, stop);
    printf("[%d]\t %g \n",gen, gputime/1000.0f);

    //CUDA - copy data from device
    if(t%2==1){
        hipMemcpy(table, d_new, N*N*sizeof(int), hipMemcpyDeviceToHost);
    }else{
        hipMemcpy(table, d_table, N*N*sizeof(int), hipMemcpyDeviceToHost);
    }
    write_to_file(table, filename, N);

    free(table);
    hipFree(d_new);
    hipFree(d_table);
    return 0;
}

/**
 * die - display an error and terminate.
 * Used when some fatal error happens
 * and continuing would mess things up.
 */
void die(const char *message){
    if(errno){
        perror(message);
    }else{
        printf("Error: %s\n", message);
    }
    exit(1);
}

/**
 * warn - display a warning and continue
 * used when something didn't go as expected
 */
void warn(const char *message){
    if(errno){
        perror(message);
    }else{
        printf("Warning: %s\n", message);
    }
    return;
}

/**
 * read_from_file - read N*N integer values from an appropriate file.
 * Saves the game's board into array X for use by other functions
 * Warns or kills the program if something goes wrong
 */
void read_from_file(int *X, char *filename, int N){

    FILE *fp = fopen(filename, "r+");
    int size = fread(X, sizeof(int), N*N, fp);
    if(!fp)
        die("Couldn't open file to read");
    if(!size)
        die("Couldn't read from file");
    if(N*N != size)
        warn("Expected to read different number of elements");

    fclose(fp);
    return;
}

/**
 * write_to_file - write N*N integer values to a binary file.
 * Saves game's board from array X to the file
 * Names the file tableNxN_new.bin, so the input file is not overwritten
 */
void write_to_file(int *X, char *filename, int N){

    //save as tableNxN_new.bin
    char newfilename[100];
    sprintf(newfilename, "cuda_table%dx%d.bin", N, N);

    FILE *fp;
    int size;
    if( ! ( fp = fopen(newfilename, "w+") ) )
        die("Couldn't open file to write");
    if( ! (size = fwrite(X, sizeof(int), N*N, fp)) )
        die("Couldn't write to file");
    if (size != N*N)
        warn("Expected to write different number of elements");

    fclose(fp);
    return;
}

/**
 * nextPower - return smallest power of 2 larger than N
 */
int nextPower(int N){
    int n=0;
    while(1){
        if(1<<n < N){
            n++;
        }else{
            return 1<<n;
        }
    }
}
